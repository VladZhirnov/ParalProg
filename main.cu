#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>
#include <time.h>

using namespace std;
using namespace std::chrono;


vector<vector<float>> readMatrix(const string &filename) {
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error: Failed to open file " << filename << endl;
        exit(1);
    }

    vector<vector<float>> matrix;
    string line;
    float num;

    while (getline(file, line)) {
        vector<float> row;
        istringstream iss(line);
        while (iss >> num) {
            row.push_back(num);
        }
        matrix.push_back(row);
    }

    file.close();
    return matrix;
}


void writeMatrix(const vector<vector<float>> &matrix, const string &filename) {
    ofstream file(filename);
    if (!file.is_open()) {
        cerr << "Error: Failed to open file " << filename << " for recording" << endl;
        exit(1);
    }

    for (const auto &row : matrix) {
        for (float val : row) {
            file << val << " ";
        }
        file << "\n";
    }
    
    file.close();
}


__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols)
    {
        float sum = 0.0f;
        for (int i = 0; i < A_cols; ++i)
        {
            sum += A[row * A_cols + i] * B[i * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}


vector<vector<float>> mulMatricesCUDA(const vector<vector<float>> &A, const vector<vector<float>> &B) {
    int A_rows = A.size(), A_cols = A[0].size(), B_cols = B[0].size();

    vector<float> h_A(A_rows * A_cols);
    vector<float> h_B(A_cols * B_cols);
    vector<float> h_C(A_rows * B_cols);

    for (int i = 0; i < A_rows; ++i)
        for (int j = 0; j < A_cols; ++j)
            h_A[i * A_cols + j] = A[i][j];

    for (int i = 0; i < A_cols; ++i)
        for (int j = 0; j < B_cols; ++j)
            h_B[i * B_cols + j] = B[i][j];

    float *d_A, *d_B, *d_C;
    size_t size_A = h_A.size() * sizeof(float);
    size_t size_B = h_B.size() * sizeof(float);
    size_t size_C = h_C.size() * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((B_cols + 15) / 16, (A_rows + 15) / 16);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, A_rows, A_cols, B_cols);
    hipDeviceSynchronize();

    hipMemcpy(h_C.data(), d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    vector<vector<float>> result(A_rows, vector<float>(B_cols));
    for (int i = 0; i < A_rows; ++i)
        for (int j = 0; j < B_cols; ++j)
            result[i][j] = h_C[i * B_cols + j];

    return result;
}


int get_time(const string &input1, const string &input2, const string &output) {
    auto start = chrono::high_resolution_clock::now();
    vector<vector<float>> vec1 = readMatrix(input1);
    vector<vector<float>> vec2 = readMatrix(input2);

    if (vec1.empty() || vec2.empty())
    {
        cerr << "Error: One or both input matrices are empty. Cannot multiply." << endl;
        return -1;
    }

    vector<vector<float>> result = mulMatricesCUDA(vec1, vec2);

    if (result.empty())
    {
        cerr << "Error: Error during multiplication" << endl;
        return -1;
    }
    writeMatrix(result, output);
    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    cout << "Times: " << duration.count() << "\n";
    return duration.count();
}


void writeTime(const string &content, const string &name_file)
{
    ofstream file;
    file.open(name_file);
    file << content;
    file.close();
}


int main() {
    int time_10 = get_time("Matrix_1/matrix1_10.txt", "Matrix_2/matrix2_10.txt", "Output/output_10.txt");
    int time_20 = get_time("Matrix_1/matrix1_20.txt", "Matrix_2/matrix2_20.txt", "Output/output_20.txt");
    int time_30 = get_time("Matrix_1/matrix1_30.txt", "Matrix_2/matrix2_30.txt", "Output/output_30.txt");
    int time_40 = get_time("Matrix_1/matrix1_40.txt", "Matrix_2/matrix2_40.txt", "Output/output_40.txt");
    int time_50 = get_time("Matrix_1/matrix1_50.txt", "Matrix_2/matrix2_50.txt", "Output/output_50.txt");
    int time_60 = get_time("Matrix_1/matrix1_60.txt", "Matrix_2/matrix2_60.txt", "Output/output_60.txt");
    int time_70 = get_time("Matrix_1/matrix1_70.txt", "Matrix_2/matrix2_70.txt", "Output/output_70.txt");
    int time_80 = get_time("Matrix_1/matrix1_80.txt", "Matrix_2/matrix2_80.txt", "Output/output_80.txt");
    int time_90 = get_time("Matrix_1/matrix1_90.txt", "Matrix_2/matrix2_90.txt", "Output/output_90.txt");
    int time_100 = get_time("Matrix_1/matrix1_100.txt", "Matrix_2/matrix2_100.txt", "Output/output_100.txt");
    int time_1000 = get_time("Matrix_1/matrix1_1000.txt", "Matrix_2/matrix2_1000.txt", "Output/output_1000.txt");
    
    stringstream total_times;
    total_times << time_10 << ", " << time_20 << ", " << time_30 << ", " << time_40 << ", " << time_50 << ", " << time_60 << ", " << time_70 << ", " << time_80 << ", " << time_90 << ", " << time_100 << ", " << time_1000;
    string times = total_times.str();
    writeTime(times, "times_CUDA.txt");
    return 0;
}